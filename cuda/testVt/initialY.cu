#include "initialY.h"
void cuGemv(float* A, float* X, float* y, int m, int n){
    float *d_X = NULL;
    float *d_A = NULL;
    float *d_y = NULL;
    const float alpha = 1;
    const float beta = 0;
    hipblasHandle_t cublasH = NULL;
    hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess;
    hipError_t cudaStat2 = hipSuccess;
    hipError_t cudaStat3 = hipSuccess;
    cudaStat1 = hipMalloc((void**)&d_A, sizeof(float) *m*n);
    cudaStat2 = hipMalloc((void**)&d_X, sizeof(float) *n);
    cudaStat3 = hipMalloc((void**)&d_y, sizeof(float) *m);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);
    cublas_status = hipblasCreate(&cublasH);
    assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

    cudaStat1 = hipMemcpy(d_A, A, sizeof(float) *m*n, hipMemcpyHostToDevice);
    cudaStat2 = hipMemcpy(d_X, X, sizeof(float) *n, hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);

    cublas_status = hipblasSgemv(
            cublasH,
            HIPBLAS_OP_N,
            m,
            n,
            &alpha,
            d_A,
            m,
            d_X,
            1,
            &beta,
            d_y,
            1
            );
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPBLAS_STATUS_SUCCESS == cublas_status);
    assert(hipSuccess == cudaStat1);

    cudaStat1 = hipMemcpy(y, d_y, sizeof(float)*m, hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);

    if (d_A    ) hipFree(d_A);
    if (d_y    ) hipFree(d_y);
    if (d_X    ) hipFree(d_X);
    if (cublasH ) hipblasDestroy(cublasH);
    hipDeviceReset();
   
}

void warmup(){
    int n = 100;
    float a[100], b[100];
    for (int i=0; i<n; i++>){
        a[i] = i;
        b[i] = i;
    }
    float *d_a, *d_b;
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipMalloc((void**)&d_a, sizeof(float)*n);
    hipMalloc((void**)&d_b, sizeof(float)*n);
    float alpha = 1.0;
    hipblasSetVector(n, sizeof(float), a, 1, d_a, 1);
    hipblasSetVector(n, sizeof(float), b, 1, d_b, 1);
    hipblasSaxpy(handle, n, &alpha, d_a,1,d_b,1);
    hipblasGetVector(n, sizeof(float), d_b, 1, a, 1);
    hipFree(d_a);
    hipFree(d_b);
    hipblasDestroy(handle);
}

#include "lsqr.h"
//solve A*x = b where size(A) m*n size(x) n*1 size(b) m*1
void lsqr(float* A, float* X, float* B, int m, int n){
    hipsolverHandle_t cusolverH = NULL;
    hipblasHandle_t cublasH = NULL;
    hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
    hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess;
    hipError_t cudaStat2 = hipSuccess;
    hipError_t cudaStat3 = hipSuccess;
    hipError_t cudaStat4 = hipSuccess;
    const int lda = m;
    const int ldb = m;
    const int nrhs = 1; // number of right hand side vectors


    /*float A[lda*m] = { 1.0, 4.0, 2.0, 2.0, 5.0, 1.0, 3.0, 6.0, 1.0};
//    float X[ldb*nrhs] = { 1.0, 1.0, 1.0}; // exact solution
    float B[ldb*nrhs] = { 6.0, 15.0, 4.0};
    float XC[ldb*nrhs]; // solution matrix from GPU
    */

    float *d_A = NULL; // linear memory of GPU
    float *d_tau = NULL; // linear memory of GPU
    float *d_B  = NULL;
    int *devInfo = NULL; // info in gpu (device copy)
    float *d_work = NULL;
    int  lwork = 0;

    const float one = 1;
// step 1: create cusolver/cublas handle
    cusolver_status = hipsolverDnCreate(&cusolverH);
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

    cublas_status = hipblasCreate(&cublasH);
    assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

// step 2: copy A and B to device
    cudaStat1 = hipMalloc ((void**)&d_A  , sizeof(float) * lda * n);
    cudaStat2 = hipMalloc ((void**)&d_tau, sizeof(float) * n);
    cudaStat3 = hipMalloc ((void**)&d_B  , sizeof(float) * ldb * nrhs);
    cudaStat4 = hipMalloc ((void**)&devInfo, sizeof(int));
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);
    assert(hipSuccess == cudaStat4);

    cudaStat1 = hipMemcpy(d_A, A, sizeof(float) * lda * n   , hipMemcpyHostToDevice);
    cudaStat2 = hipMemcpy(d_B, B, sizeof(float) * ldb * nrhs, hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);

     
// step 3: query working space of geqrf and ormqr
    cusolver_status = hipsolverDnSgeqrf_bufferSize(
        cusolverH, 
        m, 
        n, 
        d_A, 
        lda, 
        &lwork);
    assert (cusolver_status == HIPSOLVER_STATUS_SUCCESS);
 
    cudaStat1 = hipMalloc((void**)&d_work, sizeof(float)*lwork);
    assert(hipSuccess == cudaStat1);

// step 4: compute QR factorization
    cusolver_status = hipsolverDnSgeqrf(
        cusolverH, 
        m, 
        n, 
        d_A, 
        lda, 
        d_tau, 
        d_work, 
        lwork, 
        devInfo);
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
    assert(hipSuccess == cudaStat1);

    // check if QR is good or not
    //cudaStat1 = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
    //assert(hipSuccess == cudaStat1);

// step 5: compute Q^T*B
    cusolver_status= hipsolverDnSormqr(
        cusolverH, 
        HIPBLAS_SIDE_LEFT, 
        HIPBLAS_OP_T,
        m, 
        nrhs, 
        n, 
        d_A, 
        lda,
        d_tau,
        d_B,
        ldb,
        d_work,
        lwork,
        devInfo);
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
    assert(hipSuccess == cudaStat1);

// step 6: compute x = R \ Q^T*B

    cublas_status = hipblasStrsm(
         cublasH,
         HIPBLAS_SIDE_LEFT,
         HIPBLAS_FILL_MODE_UPPER,
         HIPBLAS_OP_N,
         HIPBLAS_DIAG_NON_UNIT,
         n,
         nrhs,
         &one,
         d_A,
         lda,
         d_B,
         ldb);
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPBLAS_STATUS_SUCCESS == cublas_status);
    assert(hipSuccess == cudaStat1);
    //transfer data back to CPU from GPU？
    cudaStat1 = hipMemcpy(X, d_B, sizeof(float)*n*nrhs, hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);

// free resources
    if (d_A    ) hipFree(d_A);
    if (d_tau  ) hipFree(d_tau);
    if (d_B    ) hipFree(d_B);
    if (devInfo) hipFree(devInfo);
    if (d_work ) hipFree(d_work);


    if (cublasH ) hipblasDestroy(cublasH);
    if (cusolverH) hipsolverDnDestroy(cusolverH);

    hipDeviceReset();
}

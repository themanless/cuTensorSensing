#include <iostream>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <cstdlib>

using namespace std;

void onebatch(float* A, float* B, float* C, int m, int n, int k, int batch){
    float *d_B = NULL;
    float *d_A = NULL;
    float *d_C = NULL;
    const float alpha = 1;
    const float beta = 0;
    hipblasHandle_t cublasH = NULL;
    hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess;
    hipError_t cudaStat2 = hipSuccess;
    hipError_t cudaStat3 = hipSuccess;
    cudaStat1 = hipMalloc((void**)&d_A, sizeof(float) *m*k*batch);
    cudaStat2 = hipMalloc((void**)&d_B, sizeof(float) *k*n);
    cudaStat3 = hipMalloc((void**)&d_C, sizeof(float) *m*n*batch);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);
    cublas_status = hipblasCreate(&cublasH);
    assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

    cudaStat1 = hipMemcpy(d_A, A, sizeof(float) *m*k*batch, hipMemcpyHostToDevice);
    cudaStat2 = hipMemcpy(d_B, B, sizeof(float) *n*k, hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);

    cublas_status = hipblasSgemmStridedBatched(
            cublasH,
            HIPBLAS_OP_N,
            HIPBLAS_OP_N,
            m,
            n,
            k,
            &alpha,
            d_A,
            m,
            m*k,
            d_B,
            k,
            0,
            &beta,
            d_C,
            m,
            m*n,
            batch
            );
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPBLAS_STATUS_SUCCESS == cublas_status);
    assert(hipSuccess == cudaStat1);

    cudaStat1 = hipMemcpy(C, d_C, sizeof(float)*m*n*batch, hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);

    if (d_A    ) hipFree(d_A);
    if (d_B    ) hipFree(d_B);
    if (d_C    ) hipFree(d_C);
    if (cublasH ) hipblasDestroy(cublasH);
    hipDeviceReset();

}
void printTensor(float* ar, int row, int col, int tuple){
    for (int z=0; z<tuple; z++){
        for (int j=0; j<row; j++){
            for (int i=0; i<col; i++){
                cout << ar[z*row*col+i*row+j] << " ";
            }
            cout << endl;
        }
        cout << "----------------\n";
    }
}

int main(){
    int m, n, k, batch;
    cin >> m >> n >> k >> batch;
    float* A = new float[m*k*batch];
    float* B = new float[k*n];
    float* C = new float[m*n*batch];
    for (int i=0; i<m*k*batch; i++){
        cin >> A[i];
    }
    for (int i=0; i<k*n; i++){
        cin >> B[i];
    }
    cout << "A is \n";
    printTensor(A, m, k, batch);
    cout << "B is \n";
    printTensor(B, k, n, 1);
    onebatch(A, B, C, m, n, k, batch);
    cout << "C is \n";
    printTensor(C, m, n, batch);
    delete A;
    delete C;
    delete B;
    return 0;
}

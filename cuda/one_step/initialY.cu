#include "initialY.h"
void cuGemv(float* A, float* X, float* y, int m, int n){
    float *d_X = NULL;
    float *d_A = NULL;
    float *d_y = NULL;
    const float alpha = 1;
    const float beta = 0;
    hipblasHandle_t cublasH = NULL;
    hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess;
    hipError_t cudaStat2 = hipSuccess;
    hipError_t cudaStat3 = hipSuccess;
    cudaStat1 = hipMalloc((void**)&d_A, sizeof(float) *m*n);
    cudaStat2 = hipMalloc((void**)&d_X, sizeof(float) *n);
    cudaStat3 = hipMalloc((void**)&d_y, sizeof(float) *m);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);
    cublas_status = hipblasCreate(&cublasH);
    assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

    cudaStat1 = hipMemcpy(d_A, A, sizeof(float) *m*n, hipMemcpyHostToDevice);
    cudaStat2 = hipMemcpy(d_X, X, sizeof(float) *n, hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);

    cublas_status = hipblasSgemv(
            cublasH,
            HIPBLAS_OP_N,
            m,
            n,
            &alpha,
            d_A,
            m,
            d_X,
            1,
            &beta,
            d_y,
            1
            );
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPBLAS_STATUS_SUCCESS == cublas_status);
    assert(hipSuccess == cudaStat1);

    cudaStat1 = hipMemcpy(y, d_y, sizeof(float)*m, hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);

    if (d_A    ) hipFree(d_A);
    if (d_y    ) hipFree(d_y);
    if (d_X    ) hipFree(d_X);
    if (cublasH ) hipblasDestroy(cublasH);
    hipDeviceReset();
   
}
void cuGemm(float* A, float* B, float* C, int m, int n, int k){
    float *d_B = NULL;
    float *d_A = NULL;
    float *d_C = NULL;
    const float alpha = 1;
    const float beta = 0;
    hipblasHandle_t cublasH = NULL;
    hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess;
    hipError_t cudaStat2 = hipSuccess;
    hipError_t cudaStat3 = hipSuccess;
    cudaStat1 = hipMalloc((void**)&d_A, sizeof(float) *m*k);
    cudaStat2 = hipMalloc((void**)&d_B, sizeof(float) *n*k);
    cudaStat3 = hipMalloc((void**)&d_C, sizeof(float) *m*n);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);
    cublas_status = hipblasCreate(&cublasH);
    assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

    cudaStat1 = hipMemcpy(d_A, A, sizeof(float) *m*k, hipMemcpyHostToDevice);
    cudaStat2 = hipMemcpy(d_B, B, sizeof(float) *n*k, hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);

    cublas_status = hipblasSgemm(
            cublasH,
            HIPBLAS_OP_T,
            HIPBLAS_OP_N,
            m,
            n,
            k,
            &alpha,
            d_A,
            k,
            d_B,
            k,
            &beta,
            d_C,
            m
            );
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPBLAS_STATUS_SUCCESS == cublas_status);
    assert(hipSuccess == cudaStat1);

    cudaStat1 = hipMemcpy(C, d_C, sizeof(float)*m*n, hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);

    if (d_A    ) hipFree(d_A);
    if (d_C    ) hipFree(d_C);
    if (d_B    ) hipFree(d_B);
    if (cublasH ) hipblasDestroy(cublasH);
    hipDeviceReset();
   
}

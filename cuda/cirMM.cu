#include "cirMM.h"

void cirMM(float* W, float* A, float* U, int m, int n, int r, int k, int d){
    float* Um = new float[m*k*r*k];
    for (int i=0; i<r; i++)
        for (int j=0; j<m; j++)
            for (int a=0; a<k; a++)
                for (int b=0; b<k; b++){
                    Um[(i*k+a)*m*k+j*k+b] = U[(i*m+j)*k+(b-a+k)%k];
                }
    float *d_U = NULL;
    float *d_A = NULL;
    float *d_W = NULL;
    const float alpha = 1;
    const float beta = 0;
    hipblasHandle_t cublasH = NULL;
    hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess;
    hipError_t cudaStat2 = hipSuccess;
    hipError_t cudaStat3 = hipSuccess;
    int K = m*k, M = d, N = r*k, batch = n;
    cout << "Um is  \n";
    for (int j=0; j<K; j++){
        for (int i=0; i<N; i++){
            cout << Um[i*K+j] << " ";
        }
        cout << endl;
    }
    cudaStat1 = hipMalloc((void**)&d_A, sizeof(float) *M*K*batch);
    cudaStat2 = hipMalloc((void**)&d_U, sizeof(float) *K*N);
    cudaStat3 = hipMalloc((void**)&d_W, sizeof(float) *M*N*batch);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);
    cublas_status = hipblasCreate(&cublasH);
    assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

    cudaStat1 = hipMemcpy(d_A, A, sizeof(float) *M*K*batch, hipMemcpyHostToDevice);
    cudaStat2 = hipMemcpy(d_U, Um, sizeof(float) *N*K, hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);

    cublas_status = hipblasSgemmStridedBatched(
            cublasH,
            HIPBLAS_OP_N,
            HIPBLAS_OP_N,
            M,
            N,
            K,
            &alpha,
            d_A,
            M,
            M*K,
            d_U,
            K,
            0,
            &beta,
            d_W,
            M,
            M*N,
            batch
            );
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPBLAS_STATUS_SUCCESS == cublas_status);
    assert(hipSuccess == cudaStat1);

    cudaStat1 = hipMemcpy(W, d_W, sizeof(float)*M*N*batch, hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);

    if (d_A    ) hipFree(d_A);
    if (d_W    ) hipFree(d_W);
    if (d_U    ) hipFree(d_U);
    if (cublasH ) hipblasDestroy(cublasH);
    hipDeviceReset();
}

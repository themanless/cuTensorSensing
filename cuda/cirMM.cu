#include "cirMM.h"

void cirMM(float* W, float* At, float* V, int M, int N, int K, int batch){
    float *d_V = NULL;
    float *d_At = NULL;
    float *d_W = NULL;
    const float alpha = 1;
    const float beta = 0;
    hipblasHandle_t cublasH = NULL;
    hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess;
    hipError_t cudaStat2 = hipSuccess;
    hipError_t cudaStat3 = hipSuccess;
    /*
    cout << "V is  \n";
    for (int j=0; j<K; j++){
        for (int i=0; i<N; i++){
            cout << V[i*K+j] << " ";
        }
        cout << endl;
    }
    */
    cudaStat1 = hipMalloc((void**)&d_At, sizeof(float) *M*K*batch);
    cudaStat2 = hipMalloc((void**)&d_V, sizeof(float) *K*N);
    cudaStat3 = hipMalloc((void**)&d_W, sizeof(float) *M*N*batch);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);
    cublas_status = hipblasCreate(&cublasH);
    assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

    cudaStat1 = hipMemcpy(d_At, At, sizeof(float) *M*K*batch, hipMemcpyHostToDevice);
    cudaStat2 = hipMemcpy(d_V, V, sizeof(float) *N*K, hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);

    cublas_status = hipblasSgemmStridedBatched(
            cublasH,
            HIPBLAS_OP_N,
            HIPBLAS_OP_T,
            M,
            N,
            K,
            &alpha,
            d_At,
            M,
            M*K,
            d_V,
            N,
            0,
            &beta,
            d_W,
            M,
            M*N,
            batch
            );
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPBLAS_STATUS_SUCCESS == cublas_status);
    assert(hipSuccess == cudaStat1);

    cudaStat1 = hipMemcpy(W, d_W, sizeof(float)*M*N*batch, hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);

    if (d_At    ) hipFree(d_At);
    if (d_W    ) hipFree(d_W);
    if (d_V    ) hipFree(d_V);
    if (cublasH ) hipblasDestroy(cublasH);
    hipDeviceReset();
}
